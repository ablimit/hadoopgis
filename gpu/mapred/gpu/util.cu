#include <iostream>
#include "gpu_refine.h"

using namespace std;

hipStream_t *stream = NULL;

int get_gpu_device_count(void)
{
	int count;
	hipGetDeviceCount(&count);
	return count;
}

void init_device_streams(int nr_devices)
{
	stream = new hipStream_t[nr_devices];
	if(!stream) {
		cerr <<"failed to allocate cuda streams" <<endl;
		exit(1);
	}

	for(int i = 0; i < nr_devices; i++) {
		hipSetDevice(i);
		hipStreamCreate(&stream[i]);
	}
}

void fini_device_streams(int nr_devices)
{
  for(int i = 0; i < nr_devices; i++) {
    hipStreamDestroy(stream[i]);
  }
  delete [] stream;
}

